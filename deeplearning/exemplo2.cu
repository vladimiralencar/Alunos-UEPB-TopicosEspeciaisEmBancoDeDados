
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// Número de elementos em cada vetor
#define N 2048 * 2048 

__global__ void my_kernel(int * a, int * b, int * c)
{
    // Determina a identificação de thread global exclusiva, por isso sabemos qual elemento processar
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ( tid < N ) // Certifique-se de que não inicializamos mais threads do que o necessário
        c[tid] = a[tid] + b[tid];
}

void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    std::cout << "Free = " << free << " Total = " << total <<std::endl;
}

int main()
{
    int *a, *b, *c;

    // Número total de bytes por vetor
    int size = N * sizeof (int); 

    // Aloca memória sem a necessidade de usar cudaMemcpy
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Inicializa memória
    for( int i = 0; i < N; ++i )
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    int threads_per_block = 128;
    int number_of_blocks = (N / threads_per_block) + 1;

    my_kernel <<< number_of_blocks, threads_per_block >>> ( a, b, c );

    // Espera até a GPU finalizar
    hipDeviceSynchronize(); 

    // Imprime os últimos 5 valores de c 
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    // Libera toda a nossa memória alocada
    report_gpu_mem();
    hipFree( a );
    report_gpu_mem(); 
    hipFree( b );
    report_gpu_mem(); 
    hipFree( c );
    report_gpu_mem();
}